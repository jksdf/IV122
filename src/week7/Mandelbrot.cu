
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime_api.h>

#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))

#define ull unsigned long long

__global__
void mandelbrot(ull x, ull y, int steps, uint8_t *res)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    ull size = y * x/8;
    for (ull i = index; i < size; i += stride) {
        for (int bytepos = 0; bytepos < 8; bytepos++) {
            float r0 = 3.0f * ((i*8+bytepos) % x) / x - 2.0f;
            float c0 = 2.0f * ((i*8+bytepos) / x) / y - 1.0f;
            float r = r0, c = c0;
            for (int step = 0; step < steps && r < 2 && r > -3 && c < 2 && c > -2; step++) {
                float r2 = r * r - c * c + r0;
                float c2 = 2 * r * c + c0;
                r = r2;
                c = c2;
            }
            if (r > 2 || r < -3 || c > 2 || c < -2) {
                res[i] |= 0x80 >> bytepos;
            }
        }
    }
}

int main(int argc, char **args)
{
    if (argc != 4) {
        std::cerr << "Bad params" << std::endl;
        return 1;
    }
    hipError_t error;
    int SCALE = atoi(args[1]);
    int STEPS = atoi(args[2]);
    char *OUT = args[3];
    ull X = 3 * SCALE, Y = 2*SCALE;
    if (X % 8 != 0) {
        std::cerr << "Scale has to be divisible by 8\n";
        return 1;
    }
    std::cerr << "Scale: " << SCALE << " (X = " << X << ", Y = " << Y << ")\nSteps: " << STEPS << "\nOutput file: " << OUT << std::endl;

    uint8_t *res;
    error = hipMalloc(&res, X*Y*sizeof(uint8_t) / 8);
    if (!res || error == hipErrorOutOfMemory) {
        std::cerr << "Unable to allocate " << X*Y*sizeof(uint8_t)/8 << " bytes of GPU memory" << std::endl;
        return 1;
    }
    
    uint8_t *dataHost = (uint8_t *)malloc(X * Y * sizeof(uint8_t)/8);
    if (!dataHost) {
        std::cerr << "Not enough RAM" << std::endl;
        return 1;
    }

    FILE *output = fopen(OUT, "wb");

    std::cerr << "Writing" << sizeof(uint8_t) * X * Y /8 << " bytes "<< std::endl;
    hipMemset(res, 0x0, X * Y * sizeof(uint8_t)/8);
    int blockSize = 256;
    int numBlocks = (X*Y + blockSize - 1) / blockSize;
    mandelbrot<<<numBlocks, blockSize>>>(X, Y, STEPS, res);
    hipDeviceSynchronize();
    hipMemcpy(dataHost, res, X * Y * sizeof(uint8_t)/8, hipMemcpyDeviceToHost);
    fwrite(dataHost, sizeof(uint8_t), X*Y/8, output);
    
    
    hipFree(res);
    std::cerr << "Closing" << std::endl;
    std::cout << "convert -size " << X << "x" << Y << " -depth 1 -define png:compression-strategy=3 gray:" << OUT << " res.png" << std::endl;
    fclose(output);
    hipProfilerStop(); 
    return 0;
}
