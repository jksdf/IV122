
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime_api.h>

__global__
void mandelbrot(int x, int y, int steps, uint8_t *res)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int size = x * y;
    for (int i = index; i < size; i += stride) {
        float r0 = 3.0f * (i % x) / x - 2.0f;
        float c0 = 2.0f * (i / x) / y - 1.0f;
        float r = r0, c = c0;
        for (int step = 0; step < steps && r < 2 && r > -3 && c < 2 && c > -2; step++) {
            float r2 = r * r - c * c + r0;
            float c2 = 2 * r * c + c0;
            r = r2;
            c = c2;
        }
        if (r > 2 || r < -3 || c > 2 || c < -2) {
            res[i] = 255; 
        } 
    }
}

int main(int argc, char **args)
{
    if (argc != 4) {
        std::cerr << "Bad params" << std::endl;
        return 1;
    }
    int SCALE = atoi(args[1]);
    int STEPS = atoi(args[2]);
    char *OUT = args[3];
    int X = 3 * SCALE, Y = 2*SCALE;
    std::cerr << "Scale: " << SCALE << " (X = " << X << ", Y = " << Y << ")\nSteps: " << STEPS << "\nOutput file: " << OUT << std::endl;

    uint8_t *res;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMalloc(&res, X*Y*sizeof(uint8_t));
    if (!res) {
        std::cerr << "Unable to allocate the memory" << std::endl;
        return 1;
    }
    hipMemset(res, 0, X * Y * sizeof(uint8_t));

    // initialize x and y arrays on the host
    int blockSize = 256;
    int numBlocks = (X*Y + blockSize - 1) / blockSize;
    std::cerr << "blockSize " << blockSize << "numBlocks " << numBlocks << std::endl;
    mandelbrot<<<numBlocks, blockSize>>>(X, Y, STEPS, res);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    
    uint8_t *dataHost = (uint8_t *)malloc(X * Y * sizeof(uint8_t));
    hipMemcpy(dataHost, res, X * Y * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(res);
    FILE *output = fopen(OUT, "wb");
    std::cerr << "Writing" << sizeof(uint8_t) * X * Y << " bytes "<< std::endl;
    for (unsigned y = 0; y < Y; y++) {
        for (unsigned x = 0; x < X; x++) {
            fwrite(dataHost+y*X+x, 1, 1, output);
        }
    }
//    fwrite(res, sizeof(uint8_t), X * Y, output);
    std::cerr << "Closing" << std::endl;
    std::cout << "convert -size " << X << "x" << Y << " -depth 8 gray:" << OUT << " res.png" << std::endl;
    fclose(output);
    hipProfilerStop(); 
    return 0;
}
